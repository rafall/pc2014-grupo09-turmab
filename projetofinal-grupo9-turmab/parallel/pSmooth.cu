#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

enum channels {RED, GREEN, BLUE};

#define usint unsigned short int;

#define THREADS_PER_BLOCK 1024

__global__ void smooth(usin *image, usint *smoothedImage, usint imageHeight, usint imageWidth)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int index = col + row*THREADS_PER_BLOCK;

	int k, l;

	/*If it's out of the image it doesn't need to do anything*/
	if(row > imageHeight || col > imageWidth)
		return;

	/*Smooth on a single pixel by getting the mean of a 5x5 square of pixels*/
	for(k = row - 2; k <= row + 2; k++)
	{
		for(l = col - 2; l <= col + 2; l++)
		{
			/*If it's trying to reach a pixel that's out of the image
			 *we consider it's value as the max color
			 */
			if(k < 0 || l < 0 || k >= imageHeight || l >= imageWidth)
			{
				mean += maxColor;
			}
			else
			{
				usint tInx = l + k*THREADS_PER_BLOCK;
				mean += image[tIdx];
			}	
		}
	}

	smoothedImage[index] = mean;

}

int main ()
{
	/*HOST variables*/
	usint *in[3], *out[3];
	char str[30];
	usint maxColor;
	usint imageHeight, imageWidth;
	usint numOfBlocks;
	int i;

	/*DEVICE variables*/
	usint dev_imageHeight, dev_imageWidth;
	usint *dev_in, *dev_out;

	/*Reading image header*/
	scanf("%s", str); /*PPM format*/
	scanf("%s", str); /*hash*/
	scanf("%s", str); /*name*/
	scanf("%u", &imageWidth);
	scanf("%u", &imageHeight);
	scanf("%u", &maxColor);

	/*Size of array*/
	usint size = imageHeight*imageWidth*sizeof(usint);

	/*Using the ceil function to round the number of blocks upward*/
	numOfBlocks = (usint) ceil(imageHeight*imageWidth/THREADS_PER_BLOCK);

	/*Allocating HOST memory*/
	for(i = 0; i < 3; i++)
	{
		in[i] = (int*) malloc (size);
		out[i] = (int*) malloc (size);
	}

	/*Allocating DEVICE memory*/
	hipMalloc( (void **) &dev_in, size);
	hipMalloc( (void **) &dev_out, size);

	/*Reading image file*/
	for(i = 0; i < imageHeight*imageWidth; i++)
	{
		scanf("%d", &p1);
		in[RED][i] = p1;
		scanf("%d", &p1);
		in[GREEN][i] = p1;
		scanf("%d", &p1);
		in[BLUE][i] = p1;
	}
	
	/*Copy input to device*/
	hipMemcpy(dev_imageWidth, &imageWidth, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_imageHeight, &imageHeight, size, hipMemcpyHostToDevice);

	/*One for each color channel*/
	for(i = 0; i < 3; i++)
	{
		/*Copy input to device*/
		hipMemcpy(dev_in, &in[i], size, hipMemcpyHostToDevice);
		hipMemcpy(dev_out, &out[i], size, hipMemcpyHostToDevice);

		/*The smooth kernel will run with numOfPixels/threadsPerBlock blocks and 1024 threads per block*/
	    smooth<<<numOfBlocks,THREADS_PER_BLOCK>>>(dev_in, dev_out, dev_imageHeight, dev_imageWidth);

	    /*Copy the result from DEVICE to HOST*/
	    hipMemcpy(out[i], dev_out, size, hipMemcpyDeviceToHost);
	}
	
    /*Falta criar e escrever o arquivo da imagem
	 *
	 * ....
	 *
	 */

    free(in);
    free(out);
    hipFree(dev_in);
    hipFree(dev_out);
    hipFree(dev_imageWidth);
    hipFree(dev_imageHeight);
    
    return 0;
}