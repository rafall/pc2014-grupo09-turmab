
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/*Show debug messages*/
#define _DEBUG_ 1

//enum channels {RED = 0, GREEN, BLUE};

//#define usint unsigned int;

#define THREADS_PER_BLOCK 1

__global__ void smooth(unsigned int *image, unsigned int *smoothedImage, unsigned int imageHeight, unsigned int imageWidth, unsigned int maxColor)
{
	int row = blockIdx.x*blockDim.x + threadIdx.x;
	int col = blockIdx.y*blockDim.y + threadIdx.y;
	int index = imageWidth*(threadIdx.y + blockDim.y*blockIdx.y) + threadIdx.x;
	int mean = 0;
	unsigned int tIdx = 0; /*index of pixels that form the 5x5 square*/
	int k, l;

	/*If it's out of the image it doesn't need to do anything*/
	if(row >= imageHeight || col >= imageWidth)
		return;
	/*Smooth on a single pixel by getting the mean of a 5x5 square of pixels*/
	for(k = row - 2; k <= row + 2; k++)
	{
		for(l = col - 2; l <= col + 2; l++)
		{
			/*If it's trying to reach a pixel that's out of the image
			 *we consider it's value as the max color
			 */
			if(k < 0 || l < 0 || k >= imageHeight || l >= imageWidth)
			{
				mean += maxColor;
			}
			else
			{
				tIdx = k*imageWidth + l;
				mean += image[tIdx];
			}	
		}
	}
	smoothedImage[index] = mean/25;
	if(_DEBUG_) 
		printf("Block x: %d Block y: %d Thread: %d\nrow %d col %d image[%d] : %d\n\n",blockIdx.x, blockIdx.y, threadIdx.x, row, col, index, image[tIdx]);
}

int main ()
{
	/*HOST variables*/
	//unsigned int *in[3], *out[3];
	char strFormat[4], strHash[2], strName[30];
	unsigned int maxColor;
	unsigned int imageHeight, imageWidth;
	unsigned int *red, *green, *blue, *sRed, *sGreen, *sBlue;
	
	int i, p1;

	/*DEVICE variables*/
	unsigned int *dev_in, *dev_out;

	/*Reading image header*/
	scanf("%s", strFormat); /*PPM format*/
	scanf("%s", strHash); /*hash*/
	scanf("%s", strName); /*name*/
	scanf("%u", &imageWidth);
	scanf("%u", &imageHeight);
	scanf("%u", &maxColor);

	/*Size of image array*/
	unsigned int size = imageHeight*imageWidth*sizeof(unsigned int);

	/*Allocating HOST memory*/
	for(i = 0; i < 3; i++)
	{
		//in[i] = (unsigned int*) malloc (size);
		//out[i] = (unsigned int*) malloc (size);
	}

	red = (unsigned int*) malloc (size);
	green = (unsigned int*) malloc (size);
	blue = (unsigned int*) malloc (size);
	sRed = (unsigned int*) malloc (size);
	sGreen = (unsigned int*) malloc (size);
	sBlue = (unsigned int*) malloc (size);

	/*Allocating DEVICE memory*/
	hipMalloc( (void **) &dev_in, size);
	hipMalloc( (void **) &dev_out, size);

	/*Reading image file*/
	for(i = 0; i < imageHeight*imageWidth; i++)
	{
		scanf("%d", &p1);
		red[i] = p1;
		scanf("%d", &p1);
		green[i] = p1;
		scanf("%d", &p1);
		blue[i] = p1;
	}

	int count = 0;
	if(_DEBUG_){
		printf("image:\n");
		for(i = 0; i < imageHeight*imageWidth; i++)
		{
			if(count%4 == 0)
				printf("\n");
			printf("%d ", red[i]);
			printf("%d ", green[i]);
			printf("%d ", blue[i]);
			count++;
			
		}
		printf("\n");
	}

	/*Defining blocks and grid dimensions*/
	/*1024 threads per block*/
	dim3 blockDim(1, 1); 
	dim3 gridDim(ceil(imageWidth/THREADS_PER_BLOCK), ceil(imageHeight/THREADS_PER_BLOCK));
	int dim = (imageWidth*imageHeight)/THREADS_PER_BLOCK;

	//for(i = 0; i < 3; i++)
	//{
		//if(_DEBUG_) printf("Color :%d\n", i);
		/*Copy input to device*/
		//cudaMemcpy(dev_in, in[0], size, cudaMemcpyHostToDevice);
		hipMemcpy(dev_in, red, size, hipMemcpyHostToDevice);
		/*The smooth kernel will run with numOfPixels/threadsPerBlock blocks and 1024 threads per block*/
	    smooth<<<gridDim,1>>>(dev_in, dev_out, imageHeight, imageWidth, maxColor);
	    /*Copy the result from DEVICE to HOST*/
	    //cudaMemcpy(out[0], dev_out, size, cudaMemcpyDeviceToHost);
	    hipMemcpy(sRed, dev_out, size, hipMemcpyDeviceToHost);
		
		hipMemcpy(dev_in, green, size, hipMemcpyHostToDevice);
		//cudaMemcpy(dev_in, in[1], size, cudaMemcpyHostToDevice);
	    smooth<<<gridDim,1>>>(dev_in, dev_out, imageHeight, imageWidth, maxColor);
	    /*Copy the result from DEVICE to HOST*/
	    //cudaMemcpy(out[1], dev_out, size, cudaMemcpyDeviceToHost);
	    hipMemcpy(sGreen, dev_out, size, hipMemcpyDeviceToHost);
	    
	    hipMemcpy(dev_in, blue, size, hipMemcpyHostToDevice);
	    //cudaMemcpy(dev_in, in[2], size, cudaMemcpyHostToDevice);
	    smooth<<<gridDim,1>>>(dev_in, dev_out, imageHeight, imageWidth, maxColor);
	    /*Copy the result from DEVICE to HOST*/
	    //cudaMemcpy(out[2], dev_out, size, cudaMemcpyDeviceToHost);
	    hipMemcpy(sBlue, dev_out, size, hipMemcpyDeviceToHost);
	    
	//}
	
    /*
	 *Creates and writes in a new image file
	 */
	FILE *smoothedImage;
	smoothedImage = fopen("out.ppm", "w");
	
	int j;
	fprintf(smoothedImage, "%s\n%s %s\n%u %u\n%u\n", strFormat, strHash, strName, imageWidth, imageHeight, maxColor);
	for(j = 0; j < imageWidth*imageHeight; j++)
	{
		if(count%4 == 0)
			fprintf(smoothedImage, "\n", NULL);
		fprintf(smoothedImage, "%d ",sRed[j]);
		fprintf(smoothedImage, "%d ",sGreen[j]);
		fprintf(smoothedImage, "%d ",sBlue[j]);
		count++;
	}
	
	if(0){	
		printf("smoothed image:\n");
		for(i = 0; i < imageHeight*imageWidth; i++)
		{
			if(count%4 == 0)
				printf("\n");
			printf("%d ", sRed[i]);
			printf("%d ", sGreen[i]);
			printf("%d ", sBlue[i]);
			count++;
		}
		printf("\n");
	}

    return 0;
}