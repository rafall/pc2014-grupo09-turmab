/* ----------------------------------------------------------------- */
/*  Copyright (c) 2014                                               */
/*             Fernando Noveletto Candiani         7239131,          */
/*             Marcius Leandro Junior              7277433,          */
/*             Rafael Hiroki de Figueiroa Minami   7573187,          */
/*                                                                   */
/*  This program is free software; you can redistribute it and/or    */
/*  modify it under the terms of the GNU General Public License as   */
/*  published by the Free Software Foundation; either version 3 of   */
/*  the License, or (at your option) any later version.  See the     */
/*  file LICENSE included with this distribution for more            */
/*  information.                                                     */
/*  email: fncandiani, marcius, rafahiroki @usp.br                   */
/* ----------------------------------------------------------------- */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

/*Show debug messages*/
#define _DEBUG_ 0

/*Variables used to generate errors*/
#define _FAILURE_ 0
#define _SUCCESS_ 1
#define _MEM_ERR_ 2

/*Default filter size*/
#define _FILT_SIZE_ 3

/*Value that is used to transform from microseconds to seconds.*/
#define _SEC_TO_USEC_ 1000000

/*shorter string for the unsigned short int type.*/
#define usint unsigned short int

/*Structure that will save all the image data, i.e. the RBG colors.*/
typedef struct image image;

struct image{
	usint *r, *g, *b;
};

/*Returns the difference between t0 and t1, i.e. t1-t0 in microseconds.*/
double time_diff(struct timeval t0, struct timeval t1){
	return ((double)t1.tv_sec*_SEC_TO_USEC_ + (double)t1.tv_usec)
		  -((double)t0.tv_sec*_SEC_TO_USEC_ + (double)t0.tv_usec);
}


/*The matrices that are constructed here are colums by rows, not the   */
/*usual rows by columns.*/

void allocate_img(image *img, usint cols, usint rows){

	/*Allocates the memory needed for the image*/
	img->r = (usint *) calloc(cols*rows,sizeof(usint));
	img->g = (usint *) calloc(cols*rows,sizeof(usint));
	img->b = (usint *) calloc(cols*rows,sizeof(usint));

}

void cuda_Free(image *img){

	hipFree(img->r);
	hipFree(img->g);
	hipFree(img->b);
	hipFree(img);

}

void free_img(image *img){

	free(img->r);
	free(img->g);
	free(img->b);
	free(img);

}

void localfilt(image *in, image *out, usint rows, usint cols, usint filt_size){

	usint bdr_diff = filt_size/2, x, y;
	short int i,j;
	unsigned long int sum_r, sum_g, sum_b;

	for (x = bdr_diff; x < rows+bdr_diff; x++){
		for (y = bdr_diff; y < cols+bdr_diff; y++){
			sum_r = 0; sum_g = 0; sum_b = 0;
			for(i = -bdr_diff; i <= bdr_diff; i++){
				for(j = -bdr_diff; j <= bdr_diff; j++){
/*					sum_r += in->r[y-j][x-i];*/
/*					sum_g += in->g[y-j][x-i];*/
/*					sum_b += in->b[y-j][x-i];*/
				}
			}
/*			out->r[y-bdr_diff][x-bdr_diff] = sum_r/(filt_size*filt_size);*/
/*			out->g[y-bdr_diff][x-bdr_diff] = sum_g/(filt_size*filt_size);*/
/*			out->b[y-bdr_diff][x-bdr_diff] = sum_b/(filt_size*filt_size);*/
		}
	}

/*	if(_DEBUG_) for(i=0;i<rows;++i){ for(j=0;j<cols;++j)*/
/*				printf("%hu %hu %hu\t", out->r[j][i],out->g[j][i],out->b[j][i]); printf("\n");}*/

}

int main(int argc, char** argv){

	usint i = 0, j = 0, rows = 0, cols = 0, filt_size = 0, max_val = 0;
	char format[3], img_name[500], hash[2];
	image *in = NULL, *out = NULL;

	/*Structure that is used to calculate the time that is taken to */
	/*to execute the algorithm. We use this approach instead of the */
	/*bash program /usr/bin/time -f "%e" because we can't consider  */
	/*the reading and writing time, i.e. the reading (scanf) and    */
	/*write (printf) takes the same amount of time in the parallel  */
	/*and sequential programs and since the speed up function is a  */
	/*non-linear function this will change the real value of the    */
	/*speed up. For example, if the sequential algorithm, including */
	/*the reading and writing, takes 20 seconds to calculate the    */
	/*smooth of an image and the parallel takes 15 seconds; the     */
	/*speed up here would be 1.33, but let's say the reading and    */
	/*writing takes 10 seconds, so the real computation time of each*/
	/*approach would be 10 and 5 seconds, respectively. In this case*/
	/*the speed up difference would be 0.67. If we consider that    */
	/*this algorithm can run for a long period of time this value   */
	/*would get bigger and bigger, so that is why we use this       */
	/*different approach.*/
	struct timeval t0, t1;


	/*Read from the parameters if they changed the filter size, if  */
	/*not use the default value of 5.                               */
	if(2 > argc){
		if(_DEBUG_) printf("No filter size provided, using default.\n");
		filt_size = _FILT_SIZE_;
	}else{
		filt_size = (usint) atoi(argv[1]);
	}

	/*Start reading the image as ppm from the stdin*/
	scanf("%s%s%s", format, hash, img_name);
	if(_DEBUG_) printf("%s\n%s %s\n", format, hash, img_name);

	scanf("%hu%hu%hu",&cols,&rows,&max_val);
	if(_DEBUG_) printf("%hu %hu\n%hu\n",cols,rows, max_val);

	in = (image *) malloc(sizeof(image));
	allocate_img(in, cols+filt_size-1, rows+filt_size-1);

	/*Read the matrix from the stdin.*/
	for(i = 0; i < rows; i++){
		for(j = 0; j < cols; j++){
			scanf("%hu%hu%hu",&in->r[((i+filt_size/2)*(cols+filt_size-1))+j+filt_size/2],
							  &in->g[((i+filt_size/2)*(cols+filt_size-1))+j+filt_size/2],
							  &in->b[((i+filt_size/2)*(cols+filt_size-1))+j+filt_size/2]);
		}
	}

	if(_DEBUG_) for(i=0;i<rows+filt_size-1;++i){ for(j=0;j<cols+filt_size-1;++j)
				printf("%hu %hu %hu\t", in->r[(i*(cols+filt_size/2+1))+j+filt_size/2-1],
										in->g[(i*(cols+filt_size/2+1))+j+filt_size/2-1],
										in->b[(i*(cols+filt_size/2+1))+j+filt_size/2-1]);
				printf("\n");}

	out = (image *) malloc(sizeof(image));
	allocate_img(out, cols, rows);

	/*Images that are goin to be used in the device.*/
	image *dev_in = NULL,*dev_out = NULL;

	gettimeofday(&t0, NULL);

	/*Allocates the memory needed in the device.*/
	hipMalloc( (void**)&dev_in, sizeof(image));
	hipMalloc( (void**)&dev_in->r, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(usint *));
	hipMalloc( (void**)&dev_in->g, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(usint *));
	hipMalloc( (void**)&dev_in->b, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(usint *));

	hipMalloc( (void**)&dev_out, sizeof(image));
	hipMalloc( (void**)&dev_out->r, (rows)*(cols) * sizeof(usint *));
	hipMalloc( (void**)&dev_out->g, (rows)*(cols) * sizeof(usint *));
	hipMalloc( (void**)&dev_out->b, (rows)*(cols) * sizeof(usint *));

	/*Copy the image from the host to the device (CPU -> GPU)*/
	hipMemcpy( dev_in, in, sizeof(image), hipMemcpyDeviceToHost);
	hipMemcpy( dev_in->r, in->r, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(usint *), hipMemcpyHostToDevice);
	hipMemcpy( dev_in->g, in->g, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(usint *), hipMemcpyHostToDevice);
	hipMemcpy( dev_in->b, in->b, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(usint *), hipMemcpyHostToDevice);

	//http://mc.stanford.edu/cgi-bin/images/b/ba/M02_2.pdf
	//http://mc.stanford.edu/cgi-bin/images/0/0a/M02_4.pdf
	//http://mc.stanford.edu/cgi-bin/images/5/5f/Darve_cme343_cuda_2.pdf
	//chama as funcoes assincronamente (quero fazer o mais facil)
	//se for mais facil chamar uma por vez podemos usar so uma matriz de retorno
	//dev_out que vai ser um usint de tamanho row*cols.
	//e passar uma matrix de cada vez... um cudaMalloc num usint na placa e passa tudo pra ele
	//pq os canais sao independentes
	//ver se nao vai dar pau a alocacao da estrutura.. to com medo por causa do mpi q dava bosta...

	//1 - copiar as matrizes para a GPU (nao esquecer de alocar memoria na GPU) - Done (ver se ta fundando)
	//2 - mais as flags necessarias (numero de threads, blocks, vao ser 3 grids (RGB)) - Aqui precisa definir o algoritmo q vai usar para as threads e talz
	//3 - copiar para a shared memory a parte da matriz q vai (syncthreads) - melhora no speed up (aula)
	//usar pra fazer o calculo (isso eh na funcao que vai ser executada na gpu)
	//4 - inicializar 3 kernels (funcoes q vao ser executada na GPU) um para cada
	// espectro de cor (RGB) - isso eh assincrono
	//5 - Pesquisar com vai esperar o resultado dos kernels para poder salvar a imagem
	// em disco com a copia do resultado da placa - FEITO

	//essa eh a funcao q vai executar na GPU
	//localfilt(in, out, rows, cols, filt_size);

	hipDeviceSynchronize();

	/*Copy the result from the device to the host. (GPU -> CPU)*/
	hipMemcpy(out, dev_out, sizeof(image), hipMemcpyDeviceToHost);
	hipMemcpy(out->r, dev_out->r, rows*cols*sizeof(usint), hipMemcpyDeviceToHost);
	hipMemcpy(out->g, dev_out->g, rows*cols*sizeof(usint), hipMemcpyDeviceToHost);
	hipMemcpy(out->b, dev_out->b, rows*cols*sizeof(usint), hipMemcpyDeviceToHost);

	/*Point where the timer is stopped.*/
	gettimeofday(&t1, NULL);

	fprintf(stderr,"%.0lf\n",time_diff(t0,t1));

	/*print the image as ppm to the stdout*/
	printf("%s\n%s %s\n",format, hash, img_name);
	printf("%hu %hu\n%hu\n", cols, rows, max_val);
	for( i = 0; i < rows; i++){
		for(j=0;j< cols-1; j++){
			printf("%hu %hu %hu   ", out->r[i*cols+j],out->g[i*cols+j],out->b[i*cols+j]);
		}
		printf("%hu %hu %hu", out->r[i*cols+j],out->g[i*cols+j],out->b[i*cols+j]);
		printf("\n");
	}

	free_img(in);
	free_img(out);

	cuda_Free(dev_in);
	cuda_Free(dev_out);

	return EXIT_SUCCESS;
}
