/* ----------------------------------------------------------------- */
/*  Copyright (c) 2014                                               */
/*             Fernando Noveletto Candiani         7239131,          */
/*             Marcius Leandro Junior              7277433,          */
/*             Rafael Hiroki de Figueiroa Minami   7573187,          */
/*                                                                   */
/*  This program is free software; you can redistribute it and/or    */
/*  modify it under the terms of the GNU General Public License as   */
/*  published by the Free Software Foundation; either version 3 of   */
/*  the License, or (at your option) any later version.  See the     */
/*  file LICENSE included with this distribution for more            */
/*  information.                                                     */
/*  email: fncandiani, marcius, rafahiroki @usp.br                   */
/* ----------------------------------------------------------------- */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

/*Show debug messages*/
#define _DEBUG_ 0

/*Variables used to generate errors*/
#define _FAILURE_ 0
#define _SUCCESS_ 1
#define _MEM_ERR_ 2

/*Default filter size*/
#define _FILT_SIZE_ 5
#define _SIZE_F_ 1024.0
#define _SIZE_ 1024

/*Value that is used to transform from microseconds to seconds.*/
#define _SEC_TO_USEC_ 1000000

/*shorter string for the unsigned short int type.*/
#define usint unsigned short int

/*Returns the difference between t0 and t1, i.e. t1-t0 in microseconds.*/
double time_diff(struct timeval t0, struct timeval t1){
	return ((double)t1.tv_sec*_SEC_TO_USEC_ + (double)t1.tv_usec)
		  -((double)t0.tv_sec*_SEC_TO_USEC_ + (double)t0.tv_usec);
}

__global__ void col_add( int *img, int *cols_sum, int rows, int cols ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < (rows)*(cols+4)){ 
		cols_sum[tid] = img[tid]+
					img[tid+(cols+4)]+
					img[tid+(2*(cols+4))]+
					img[tid+(3*(cols+4))]+
					img[tid+(4*(cols+4))];
	}
}

__global__ void smooth(int *sum,int *out,int rows, int cols, int *c){

	int tid = blockIdx.x * blockDim.x + threadIdx.x ; // cria id único para cada thread

	if(tid < rows*cols){ // faz as somas necessárias
		out[tid] = sum[tid+2+(4*(tid/cols))-2] + 
				sum[tid+2+(4*(tid/cols))-1] + 
				sum[tid+2+(4*(tid/cols))] +
				sum[tid+2+(4*(tid/cols))+1] +
				sum[tid+2+(4*(tid/cols))+2] ;
		
		// divide a soma pelo tamanha da mascara
		out[tid] /= _FILT_SIZE_*_FILT_SIZE_;

	}
}

int main(int argc, char** argv){

	usint i = 0, j = 0, rows = 0, cols = 0, filt_size = 0, max_val = 0;
	char format[3], img_name[500], hash[2];
	int *in_r = NULL, *in_g = NULL, *in_b = NULL, *out_r = NULL, *out_g = NULL, *out_b = NULL;

	/*Structure that is used to calculate the time that is taken to */
	/*to execute the algorithm. We use this approach instead of the */
	/*bash program /usr/bin/time -f "%e" because we can't consider  */
	/*the reading and writing time, i.e. the reading (scanf) and    */
	/*write (printf) takes the same amount of time in the parallel  */
	/*and sequential programs and since the speed up function is a  */
	/*non-linear function this will change the real value of the    */
	/*speed up. For example, if the sequential algorithm, including */
	/*the reading and writing, takes 20 seconds to calculate the    */
	/*smooth of an image and the parallel takes 15 seconds; the     */
	/*speed up here would be 1.33, but let's say the reading and    */
	/*writing takes 10 seconds, so the real computation time of each*/
	/*approach would be 10 and 5 seconds, respectively. In this case*/
	/*the speed up difference would be 0.67. If we consider that    */
	/*this algorithm can run for a long period of time this value   */
	/*would get bigger and bigger, so that is why we use this       */
	/*different approach.*/
	struct timeval t0, t1;

	/*Read from the parameters if they changed the filter size, if  */
	/*not use the default value of 5.                               */
	if(2 > argc){
		if(_DEBUG_) printf("No filter size provided, using default.\n");
		filt_size = _FILT_SIZE_;
	}else{
		//filt_size = (usint) atoi(argv[1]);
		filt_size = _FILT_SIZE_;
	}

	/*Start reading the image as ppm from the stdin*/
	scanf("%s%s%s", format, hash, img_name);
	if(_DEBUG_) printf("%s\n%s %s\n", format, hash, img_name);

	scanf("%hu%hu%hu",&cols,&rows,&max_val);
	if(_DEBUG_) printf("%hu %hu\n%hu\n",cols,rows, max_val);

	in_r = (int *) calloc((cols+4)*(rows+4),sizeof(usint));
	in_g = (int *) calloc((cols+4)*(rows+4),sizeof(usint));
	in_b = (int *) calloc((cols+4)*(rows+4),sizeof(usint));

	/*Read the matrix from the stdin.*/
	for(i = 0; i < rows; i++){
		for(j = 0; j < cols; j++){
			scanf("%d%d%d",&in_r[((i+filt_size/2)*(cols+filt_size-1))+j+filt_size/2],
							  &in_g[((i+filt_size/2)*(cols+filt_size-1))+j+filt_size/2],
							  &in_b[((i+filt_size/2)*(cols+filt_size-1))+j+filt_size/2]);
		}
	}

	if(_DEBUG_) for(i=0;i<rows+filt_size-1;++i){ for(j=0;j<cols+filt_size-1;++j)
				printf("%d %d %d\t", in_r[(i*(cols+filt_size/2+1))+j+filt_size/2-1],
										in_g[(i*(cols+filt_size/2+1))+j+filt_size/2-1],
										in_b[(i*(cols+filt_size/2+1))+j+filt_size/2-1]);
				printf("\n");}

	out_r = (int *) calloc(cols*rows,sizeof(usint));
	out_g = (int *) calloc(cols*rows,sizeof(usint));
	out_b = (int *) calloc(cols*rows,sizeof(usint));

	/*Images that are goin to be used in the device.*/
	int *dev_in = NULL, *dev_out = NULL;
	int *sum, *c;

	float numblocks;
	int blocks;

	if(cols > rows){
		numblocks = (cols/_SIZE_F_)*rows;
	}
	else{
		numblocks = (rows/_SIZE_F_)*cols;
	}

	blocks = (int) (numblocks + 1.0);


	gettimeofday(&t0, NULL);

	/*Allocates the memory needed in the device.*/
	hipMalloc( (void**)&dev_in, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(int));
	hipMalloc( (void**)&dev_out, (rows)*(cols) * sizeof(int));
	hipMalloc( (void**)&sum,(rows*(cols+4))*sizeof(int));
	hipMalloc( (void**)&c,(1*sizeof(int)));

	/*Copy the image from the host to the device (CPU -> GPU)*/
	hipMemcpy( dev_in, in_r, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(int), hipMemcpyHostToDevice);
	col_add<<<blocks,_SIZE_>>>(dev_in,sum,rows,cols);
	smooth<<<blocks,_SIZE_>>>(sum,dev_out,rows,cols,c);
	hipMemcpy(out_r, dev_out, rows*cols*sizeof(int), hipMemcpyDeviceToHost);
	
	/*Copy the image from the host to the device (CPU -> GPU)*/
	hipMemcpy( dev_in, in_g, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(int), hipMemcpyHostToDevice);
	col_add<<<blocks,_SIZE_>>>(dev_in,sum,rows,cols);
	smooth<<<blocks,_SIZE_>>>(sum,dev_out,rows,cols,c);
	hipMemcpy(out_g, dev_out, rows*cols*sizeof(int), hipMemcpyDeviceToHost);

	/*Copy the image from the host to the device (CPU -> GPU)*/
	hipMemcpy( dev_in, in_b, (rows+filt_size-1)*(cols+filt_size-1)*sizeof(int), hipMemcpyHostToDevice);
	col_add<<<blocks,_SIZE_>>>(dev_in,sum,rows,cols);
	smooth<<<blocks,_SIZE_>>>(sum,dev_out,rows,cols,c);
	hipMemcpy(out_b, dev_out, rows*cols*sizeof(int), hipMemcpyDeviceToHost);

//	cudaDeviceSynchronize();

	/*Copy the result from the device to the host. (GPU -> CPU)*/

	/*Point where the timer is stopped.*/
	gettimeofday(&t1, NULL);

	fprintf(stderr,"%.0lf\n",time_diff(t0,t1));

	/*print the image as ppm to the stdout*/
	printf("%s\n%s %s\n",format, hash, img_name);
	printf("%hu %hu\n%hu\n", cols, rows, max_val);
	for( i = 0; i < rows; i++){
		for(j=0;j< cols-1; j++){
			printf("%d %d %d   ", out_r[i*cols+j],out_g[i*cols+j],out_b[i*cols+j]);
		}
		printf("%d %d %d	", out_r[i*cols+j],out_g[i*cols+j],out_b[i*cols+j]);
		printf("\n");
	}


	free(in_r);
	free(in_g);
	free(in_b);

	free(out_r);
	free(out_g);
	free(out_b);

	hipFree(dev_in);
	hipFree(dev_out);
	hipFree(sum);

	return EXIT_SUCCESS;
}
